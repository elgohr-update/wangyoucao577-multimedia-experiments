

#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel function that will be executed on GPU
__global__ void add(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

int main() {

  const int kThreadsPerBlock = 512;
  const int kN = 2048 * 2048;
  int *a, *b, *c;       // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = kN * sizeof(int);

  // allocate space for host copies of a, b, c
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);

  // set input
  for (int i = 0; i < kN; i++) {
    a[i] = i;
    b[i] = 2 * i;
  }

  // allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() on GPU
  add<<<kN / kThreadsPerBlock, kThreadsPerBlock>>>(d_a, d_b, d_c);

  // Copy back to host
  auto err = hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("hipMemcpy failed, err %s(%d)\n", hipGetErrorString(err), err);
  }

  // print
  printf("Hello world, Vector add!\n");
  for (int i = 0; i < kN; i++) {
    if (i == 0 || i == kN - 1) {
      // only print first and last one
      printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
  }

  // Free
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(a);
  free(b);
  free(c);

  return 0;
}
